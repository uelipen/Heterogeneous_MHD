#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "hip/hip_runtime.h"

#include "parameter.h" 
#include "array_definition.h"
#include "cuda_funclist.h"

__global__ void transpose13_kernel(float *t13_ut, float *t13_bt, float *t13_u, float *t13_b, int *t13_nx, int *t13_ny, int *t13_nz);
__global__ void transpose12_kernel(float *t12_ut, float *t12_bt, float *t12_u, float *t12_b, int *t12_nx, int *t12_ny, int *t12_nz);
__global__ void transpose13_nz1_kernel(float *t13_nz1_ut, float *t13_nz1_bt, float *t13_nz1_u, float *t13_nz1_b, int *t13_nz1_nx, int *t13_nz1_ny, int *t13_nz1_nz);
__global__ void transpose13_nx1_kernel(float *t13_nx1_ut, float *t13_nx1_bt, float *t13_nx1_u, float *t13_nx1_b, int *t13_nx1_nx, int *t13_nx1_ny, int *t13_nx1_nz);

void cuda_transpose13(float *t13_ut, float *t13_bt, float *t13_u, float *t13_b, int *t13_nx, int *t13_ny, int *t13_nz, int *h_t13_nx, int *h_t13_ny, int *h_t13_nz)
{
//      initialization
int Totalthreads = (*h_t13_nx)*(*h_t13_ny)*(*h_t13_nz);
int numThreadsPerBlock = *h_t13_nx;
int numBlocks = Totalthreads/numThreadsPerBlock;
int NumOfU = 5;
int NumOfB = 3;
size_t u_memSize = NumOfU * numBlocks * numThreadsPerBlock * sizeof(float);
size_t b_memSize = NumOfB * numBlocks * numThreadsPerBlock * sizeof(float);
//      send it to device to calculate
dim3 dimGrid(*h_t13_ny,*h_t13_nz);
dim3 dimBlock(*h_t13_nx);
if ((*h_t13_nx)==(*h_t13_nz))
{
        transpose13_kernel<<< dimGrid, dimBlock >>>(t13_ut,t13_bt,t13_u,t13_b,t13_nx,t13_ny,t13_nz);
}
else if ((*h_t13_nz)==1)
{
        transpose13_nz1_kernel<<< dimGrid, dimBlock >>>(t13_ut,t13_bt,t13_u,t13_b,t13_nx,t13_ny,t13_nz);
}
else if ((*h_t13_nx)==1)
{
        transpose13_nx1_kernel<<< dimGrid, dimBlock >>>(t13_ut,t13_bt,t13_u,t13_b,t13_nx,t13_ny,t13_nz);
}
else
{
        printf("nz<>nx not supported\n");
}
//
hipDeviceSynchronize();
//
checkCUDAError("kernel execution in cuda_transpose13");
//      hipMemcpy
//      from d_ut to d_u, in device
//      from d_bt to d_b, in device
hipMemcpy(t13_u,t13_ut, u_memSize, hipMemcpyDeviceToDevice );
hipMemcpy(t13_b,t13_bt, b_memSize, hipMemcpyDeviceToDevice );
//
checkCUDAError("memcpy: from device to device, in cuda_transpose13");
//
}

__global__ void transpose13_kernel(float *t13_ut, float *t13_bt, float *t13_u, float *t13_b, int *t13_nx, int *t13_ny, int *t13_nz)
{
/*
two dimensional array of blocks on grid where each block has one dimensional array of threads:
UniqueBlockIndex = blockIdx.y * gridDim.x + blockIdx.x;
UniqueThreadIndex = UniqueBlockIndex * blockDim.x + threadIdx.x;
*/
/*
i = threadIdx.x
j = blockIdx.x
k = blockIdx.y
nx = blockDim.x
ny = gridDim.x
nz = gridDim.y
*/
t13_ut[a4D_FinC(5,gridDim.y,gridDim.x,blockDim.x,(1-1),blockIdx.y,blockIdx.x,threadIdx.x)]=t13_u[a4D_FinC(5,blockDim.x,gridDim.x,gridDim.y,(1-1),threadIdx.x,blockIdx.x,blockIdx.y)];
t13_ut[a4D_FinC(5,gridDim.y,gridDim.x,blockDim.x,(2-1),blockIdx.y,blockIdx.x,threadIdx.x)]=t13_u[a4D_FinC(5,blockDim.x,gridDim.x,gridDim.y,(4-1),threadIdx.x,blockIdx.x,blockIdx.y)];
t13_ut[a4D_FinC(5,gridDim.y,gridDim.x,blockDim.x,(3-1),blockIdx.y,blockIdx.x,threadIdx.x)]=t13_u[a4D_FinC(5,blockDim.x,gridDim.x,gridDim.y,(3-1),threadIdx.x,blockIdx.x,blockIdx.y)];
t13_ut[a4D_FinC(5,gridDim.y,gridDim.x,blockDim.x,(4-1),blockIdx.y,blockIdx.x,threadIdx.x)]=t13_u[a4D_FinC(5,blockDim.x,gridDim.x,gridDim.y,(2-1),threadIdx.x,blockIdx.x,blockIdx.y)];
t13_ut[a4D_FinC(5,gridDim.y,gridDim.x,blockDim.x,(5-1),blockIdx.y,blockIdx.x,threadIdx.x)]=t13_u[a4D_FinC(5,blockDim.x,gridDim.x,gridDim.y,(5-1),threadIdx.x,blockIdx.x,blockIdx.y)];
//
t13_bt[a4D_FinC(3,gridDim.y,gridDim.x,blockDim.x,(1-1),blockIdx.y,blockIdx.x,threadIdx.x)]=t13_b[a4D_FinC(3,blockDim.x,gridDim.x,gridDim.y,(3-1),threadIdx.x,blockIdx.x,blockIdx.y)];
t13_bt[a4D_FinC(3,gridDim.y,gridDim.x,blockDim.x,(2-1),blockIdx.y,blockIdx.x,threadIdx.x)]=t13_b[a4D_FinC(3,blockDim.x,gridDim.x,gridDim.y,(2-1),threadIdx.x,blockIdx.x,blockIdx.y)];
t13_bt[a4D_FinC(3,gridDim.y,gridDim.x,blockDim.x,(3-1),blockIdx.y,blockIdx.x,threadIdx.x)]=t13_b[a4D_FinC(3,blockDim.x,gridDim.x,gridDim.y,(1-1),threadIdx.x,blockIdx.x,blockIdx.y)];
return;
}

__global__ void transpose13_nz1_kernel(float *t13_nz1_ut, float *t13_nz1_bt, float *t13_nz1_u, float *t13_nz1_b, int *t13_nz1_nx, int *t13_nz1_ny, int *t13_nz1_nz)
{
/*
i = threadIdx.x
j = blockIdx.x
k = blockIdx.y
nx = blockDim.x
ny = gridDim.x
nz = gridDim.y
*/
//	transpose12
t13_nz1_ut[a4D_FinC(5,gridDim.x,blockDim.x,gridDim.y,(1-1),blockIdx.x,threadIdx.x,blockIdx.y)]=t13_nz1_u[a4D_FinC(5,blockDim.x,gridDim.x,gridDim.y,(1-1),threadIdx.x,blockIdx.x,blockIdx.y)];
t13_nz1_ut[a4D_FinC(5,gridDim.x,blockDim.x,gridDim.y,(2-1),blockIdx.x,threadIdx.x,blockIdx.y)]=t13_nz1_u[a4D_FinC(5,blockDim.x,gridDim.x,gridDim.y,(3-1),threadIdx.x,blockIdx.x,blockIdx.y)];
t13_nz1_ut[a4D_FinC(5,gridDim.x,blockDim.x,gridDim.y,(3-1),blockIdx.x,threadIdx.x,blockIdx.y)]=t13_nz1_u[a4D_FinC(5,blockDim.x,gridDim.x,gridDim.y,(2-1),threadIdx.x,blockIdx.x,blockIdx.y)];
t13_nz1_ut[a4D_FinC(5,gridDim.x,blockDim.x,gridDim.y,(4-1),blockIdx.x,threadIdx.x,blockIdx.y)]=t13_nz1_u[a4D_FinC(5,blockDim.x,gridDim.x,gridDim.y,(4-1),threadIdx.x,blockIdx.x,blockIdx.y)];
t13_nz1_ut[a4D_FinC(5,gridDim.x,blockDim.x,gridDim.y,(5-1),blockIdx.x,threadIdx.x,blockIdx.y)]=t13_nz1_u[a4D_FinC(5,blockDim.x,gridDim.x,gridDim.y,(5-1),threadIdx.x,blockIdx.x,blockIdx.y)];
//
t13_nz1_bt[a4D_FinC(3,gridDim.x,blockDim.x,gridDim.y,(1-1),blockIdx.x,threadIdx.x,blockIdx.y)]=t13_nz1_b[a4D_FinC(3,blockDim.x,gridDim.x,gridDim.y,(2-1),threadIdx.x,blockIdx.x,blockIdx.y)];
t13_nz1_bt[a4D_FinC(3,gridDim.x,blockDim.x,gridDim.y,(2-1),blockIdx.x,threadIdx.x,blockIdx.y)]=t13_nz1_b[a4D_FinC(3,blockDim.x,gridDim.x,gridDim.y,(1-1),threadIdx.x,blockIdx.x,blockIdx.y)];
t13_nz1_bt[a4D_FinC(3,gridDim.x,blockDim.x,gridDim.y,(3-1),blockIdx.x,threadIdx.x,blockIdx.y)]=t13_nz1_b[a4D_FinC(3,blockDim.x,gridDim.x,gridDim.y,(3-1),threadIdx.x,blockIdx.x,blockIdx.y)];
//
//	second part
float temp1,temp2,temp3;
temp1=t13_nz1_ut[a4D_FinC(5,1,gridDim.x,blockDim.x,(2-1),(1-1),blockIdx.x,threadIdx.x)];
temp2=t13_nz1_ut[a4D_FinC(5,1,gridDim.x,blockDim.x,(3-1),(1-1),blockIdx.x,threadIdx.x)];
temp3=t13_nz1_ut[a4D_FinC(5,1,gridDim.x,blockDim.x,(4-1),(1-1),blockIdx.x,threadIdx.x)];
t13_nz1_ut[a4D_FinC(5,1,gridDim.x,blockDim.x,(2-1),(1-1),blockIdx.x,threadIdx.x)]=temp3;
t13_nz1_ut[a4D_FinC(5,1,gridDim.x,blockDim.x,(3-1),(1-1),blockIdx.x,threadIdx.x)]=temp1;
t13_nz1_ut[a4D_FinC(5,1,gridDim.x,blockDim.x,(4-1),(1-1),blockIdx.x,threadIdx.x)]=temp2;
//
temp1=t13_nz1_bt[a4D_FinC(3,1,gridDim.x,blockDim.x,(1-1),(1-1),blockIdx.x,threadIdx.x)];
temp2=t13_nz1_bt[a4D_FinC(3,1,gridDim.x,blockDim.x,(2-1),(1-1),blockIdx.x,threadIdx.x)];
temp3=t13_nz1_bt[a4D_FinC(3,1,gridDim.x,blockDim.x,(3-1),(1-1),blockIdx.x,threadIdx.x)];
t13_nz1_bt[a4D_FinC(3,1,gridDim.x,blockDim.x,(1-1),(1-1),blockIdx.x,threadIdx.x)]=temp3;
t13_nz1_bt[a4D_FinC(3,1,gridDim.x,blockDim.x,(1-1),(2-1),blockIdx.x,threadIdx.x)]=temp1;
t13_nz1_bt[a4D_FinC(3,1,gridDim.x,blockDim.x,(1-1),(3-1),blockIdx.x,threadIdx.x)]=temp2;
//
return;
}

__global__ void transpose13_nx1_kernel(float *t13_nx1_ut, float *t13_nx1_bt, float *t13_nx1_u, float *t13_nx1_b, int *t13_nx1_nx, int *t13_nx1_ny, int *t13_nx1_nz)
{
/*
i = threadIdx.x
j = blockIdx.x
k = blockIdx.y
nx = blockDim.x
ny = gridDim.x
nz = gridDim.y
*/
//      transpose12
t13_nx1_ut[a4D_FinC(5,gridDim.y,gridDim.x,blockDim.x,(1-1),blockIdx.y,blockIdx.x,threadIdx.x)]=t13_nx1_u[a4D_FinC(5,gridDim.x,gridDim.y,blockDim.x,(1-1),blockIdx.x,blockIdx.y,threadIdx.x)];
t13_nx1_ut[a4D_FinC(5,gridDim.y,gridDim.x,blockDim.x,(2-1),blockIdx.y,blockIdx.x,threadIdx.x)]=t13_nx1_u[a4D_FinC(5,gridDim.x,gridDim.y,blockDim.x,(3-1),blockIdx.x,blockIdx.y,threadIdx.x)];
t13_nx1_ut[a4D_FinC(5,gridDim.y,gridDim.x,blockDim.x,(3-1),blockIdx.y,blockIdx.x,threadIdx.x)]=t13_nx1_u[a4D_FinC(5,gridDim.x,gridDim.y,blockDim.x,(2-1),blockIdx.x,blockIdx.y,threadIdx.x)];
t13_nx1_ut[a4D_FinC(5,gridDim.y,gridDim.x,blockDim.x,(4-1),blockIdx.y,blockIdx.x,threadIdx.x)]=t13_nx1_u[a4D_FinC(5,gridDim.x,gridDim.y,blockDim.x,(4-1),blockIdx.x,blockIdx.y,threadIdx.x)];
t13_nx1_ut[a4D_FinC(5,gridDim.y,gridDim.x,blockDim.x,(5-1),blockIdx.y,blockIdx.x,threadIdx.x)]=t13_nx1_u[a4D_FinC(5,gridDim.x,gridDim.y,blockDim.x,(5-1),blockIdx.x,blockIdx.y,threadIdx.x)];
//
t13_nx1_bt[a4D_FinC(3,gridDim.y,gridDim.x,blockDim.x,(1-1),blockIdx.y,blockIdx.x,threadIdx.x)]=t13_nx1_b[a4D_FinC(3,gridDim.x,gridDim.y,blockDim.x,(2-1),blockIdx.x,blockIdx.y,threadIdx.x)];
t13_nx1_bt[a4D_FinC(3,gridDim.y,gridDim.x,blockDim.x,(2-1),blockIdx.y,blockIdx.x,threadIdx.x)]=t13_nx1_b[a4D_FinC(3,gridDim.x,gridDim.y,blockDim.x,(1-1),blockIdx.x,blockIdx.y,threadIdx.x)];
t13_nx1_bt[a4D_FinC(3,gridDim.y,gridDim.x,blockDim.x,(3-1),blockIdx.y,blockIdx.x,threadIdx.x)]=t13_nx1_b[a4D_FinC(3,gridDim.x,gridDim.y,blockDim.x,(3-1),blockIdx.x,blockIdx.y,threadIdx.x)];
//
//	second part
float temp1,temp2,temp3;
temp1=t13_nx1_ut[a4D_FinC(5,gridDim.y,gridDim.x,1,(2-1),blockIdx.y,blockIdx.x,(1-1))];
temp2=t13_nx1_ut[a4D_FinC(5,gridDim.y,gridDim.x,1,(3-1),blockIdx.y,blockIdx.x,(1-1))];
temp3=t13_nx1_ut[a4D_FinC(5,gridDim.y,gridDim.x,1,(4-1),blockIdx.y,blockIdx.x,(1-1))];
t13_nx1_ut[a4D_FinC(5,gridDim.y,gridDim.x,1,(2-1),blockIdx.y,blockIdx.x,(1-1))]=temp3;
t13_nx1_ut[a4D_FinC(5,gridDim.y,gridDim.x,1,(3-1),blockIdx.y,blockIdx.x,(1-1))]=temp1;
t13_nx1_ut[a4D_FinC(5,gridDim.y,gridDim.x,1,(4-1),blockIdx.y,blockIdx.x,(1-1))]=temp2;
//
temp1=t13_nx1_bt[a4D_FinC(3,gridDim.y,gridDim.x,1,(1-1),blockIdx.y,blockIdx.x,(1-1))];
temp2=t13_nx1_bt[a4D_FinC(3,gridDim.y,gridDim.x,1,(2-1),blockIdx.y,blockIdx.x,(1-1))];
temp3=t13_nx1_bt[a4D_FinC(3,gridDim.y,gridDim.x,1,(3-1),blockIdx.y,blockIdx.x,(1-1))];
t13_nx1_bt[a4D_FinC(3,gridDim.y,gridDim.x,1,(1-1),blockIdx.y,blockIdx.x,(1-1))]=temp3;
t13_nx1_bt[a4D_FinC(3,gridDim.y,gridDim.x,1,(2-1),blockIdx.y,blockIdx.x,(1-1))]=temp1;
t13_nx1_bt[a4D_FinC(3,gridDim.y,gridDim.x,1,(3-1),blockIdx.y,blockIdx.x,(1-1))]=temp2;
//
return;
}
