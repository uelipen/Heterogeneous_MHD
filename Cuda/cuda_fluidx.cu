#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "hip/hip_runtime.h"

#include "parameter.h" 
#include "array_definition.h"
#include "cuda_funclist.h"

__global__ void fluidx_kernel(float *flu_u, float *flu_b, int *flu_nx, int *flu_ny, int *flu_nz, float *flu_dt);
__device__ void mhdflux(float *mf_v, float *mf_c, float *mf_u, float *mf_b, int *mf_n);

void cuda_fluidx(float *fluidx_u, float *fluidx_b, int *fluidx_nx, int *fluidx_ny, int *fluidx_nz, float *fluidx_dt, int *h_fluidx_nx, int *h_fluidx_ny, int *h_fluidx_nz)
{
//      send it to device to calculate
dim3 dimGrid(*h_fluidx_ny,*h_fluidx_nz);
dim3 dimBlock(*h_fluidx_nx);
fluidx_kernel<<< dimGrid, dimBlock >>>( fluidx_u, fluidx_b, fluidx_nx, fluidx_ny, fluidx_nz, fluidx_dt);
//
hipDeviceSynchronize();
//
checkCUDAError("kernel execution in cuda_fluidx");
//
}

__global__ void fluidx_kernel(float *flu_u, float *flu_b, int *flu_nx, int *flu_ny, int *flu_nz, float *flu_dt)
{
/*
two dimensional array of blocks on grid where each block has one dimensional array of threads:
UniqueBlockIndex = blockIdx.y * gridDim.x + blockIdx.x;
UniqueThreadIndex = UniqueBlockIndex * blockDim.x + threadIdx.x;
*/
//
__shared__ float flu_s_b[3*BLOCK_SIZE];
__shared__ float flu_s_u[5*BLOCK_SIZE];
__shared__ float flu_s_jp_b2[BLOCK_SIZE];
__shared__ float flu_s_kp_b3[BLOCK_SIZE];
//
for (int ii=0; ii<3; ii++)
{
	flu_s_b[a2D_FinC(3,blockDim.x,ii,threadIdx.x)]=flu_b[a4D_FinC(3,blockDim.x,gridDim.x,gridDim.y,ii,threadIdx.x,blockIdx.x,blockIdx.y)];
}
//
for (int ii=0; ii<5; ii++)
{
	flu_s_u[a2D_FinC(5,blockDim.x,ii,threadIdx.x)]=flu_u[a4D_FinC(5,blockDim.x,gridDim.x,gridDim.y,ii,threadIdx.x,blockIdx.x,blockIdx.y)];
}
//
int flu_jp,flu_kp;
flu_jp=(blockIdx.x+1)%(*flu_ny);
flu_kp=(blockIdx.y+1)%(*flu_nz);
flu_s_jp_b2[threadIdx.x]=flu_b[a4D_FinC(3,blockDim.x,gridDim.x,gridDim.y,(2-1),threadIdx.x,flu_jp,blockIdx.y)];
flu_s_kp_b3[threadIdx.x]=flu_b[a4D_FinC(3,blockDim.x,gridDim.x,gridDim.y,(3-1),threadIdx.x,blockIdx.x,flu_kp)];
//
/*
i = threadIdx.x
j = blockIdx.x
k = blockIdx.y 
nx = blockDim.x
ny = gridDim.x
nz = gridDim.y
*/
__shared__ float flu_s_b3x[3*BLOCK_SIZE];
for (int ii=0; ii<3; ii++)
{
	flu_s_b3x[a2D_FinC(3,blockDim.x,ii,threadIdx.x)]=flu_s_b[a2D_FinC(3,blockDim.x,ii,threadIdx.x)]/2.0;
}
__syncthreads();
//
int flu_imp,flu_imm;
flu_imm=(threadIdx.x+(*flu_nx)-1)%(*flu_nx);
flu_imp=(threadIdx.x+1)%(*flu_nx);
float flu_temp[3];
flu_temp[(1-1)]=flu_s_b3x[a2D_FinC(3,blockDim.x,(1-1),flu_imp)];
flu_temp[(2-1)]=flu_s_jp_b2[threadIdx.x]/2.0;
flu_temp[(3-1)]=flu_s_kp_b3[threadIdx.x]/2.0;
__syncthreads();
for (int ii=0; ii<3; ii++)
{
	flu_s_b3x[a2D_FinC(3,blockDim.x,ii,threadIdx.x)]=flu_s_b3x[a2D_FinC(3,blockDim.x,ii,threadIdx.x)]+flu_temp[ii];
}
__syncthreads();
//
// --- tvd1 part
//	first mhdflux
float tvd1_u[5], tvd1_b[3];
for (int ii=0; ii<5; ii++)
{
	tvd1_u[ii]=flu_s_u[a2D_FinC(5,blockDim.x,ii,threadIdx.x)];
}
for (int ii=0; ii<3; ii++)
{
	tvd1_b[ii]=flu_s_b3x[a2D_FinC(3,blockDim.x,ii,threadIdx.x)];
}
//
float v[5];
float thread_c;
int thread_n;
thread_n=blockDim.x;
mhdflux(v,&thread_c,tvd1_u,tvd1_b,&thread_n);
//
__shared__ float mhdflux_max[BLOCK_SIZE];
mhdflux_max[threadIdx.x]=thread_c;
__shared__ float s_c[BLOCK_SIZE];
__syncthreads();
if (threadIdx.x==0)
{
float temp_c_max;
temp_c_max=0.0;
for (int i=0; i<BLOCK_SIZE; i++)
{
	if (mhdflux_max[i]>temp_c_max) temp_c_max=mhdflux_max[i];
}
for (int i=0; i<BLOCK_SIZE; i++)
{
	s_c[i]=temp_c_max;
}
}
__syncthreads();
//__shared__ float c;
float c;
c=s_c[threadIdx.x];
//
if (c>0)
{
	for (int ii=0; ii<5; ii++)
	{
		v[ii]=v[ii]/c;
	}
}
// --- tvd1 part 1
float wr[5];
for (int ii=0; ii<5; ii++)
{
	wr[ii]=tvd1_u[ii]+v[ii];
}
float wl[5];
for (int ii=0; ii<5; ii++)
{
	wl[ii]=tvd1_u[ii]-v[ii];
}
float fr[5];
for (int ii=0; ii<5; ii++)
{
	fr[ii]=c*wr[ii];
}
__shared__ float tvd1_s_tmp1[5*BLOCK_SIZE];
for (int ii=0; ii<5; ii++)
{
	tvd1_s_tmp1[a2D_FinC(5,blockDim.x,ii,threadIdx.x)]=wl[ii];
}
__syncthreads();
float fl[5];
for (int ii=0; ii<5; ii++)
{
	fl[ii]=c*tvd1_s_tmp1[a2D_FinC(5,blockDim.x,ii,flu_imp)];
}
__syncthreads();
//
float flux[5];
for (int ii=0; ii<5; ii++)
{
	flux[ii]=(fr[ii]-fl[ii])/2.0;
}
//
__shared__ float tvd1_s_tmp2[5*BLOCK_SIZE];
for (int ii=0; ii<5; ii++)
{
	tvd1_s_tmp2[a2D_FinC(5,blockDim.x,ii,threadIdx.x)]=flux[ii];
}
__syncthreads();
float tvd1_u1[5];
for (int ii=0; ii<5; ii++)
{
	tvd1_u1[ii]=tvd1_u[ii]-(flux[ii]-tvd1_s_tmp2[a2D_FinC(5,blockDim.x,ii,flu_imm)])*(*flu_dt)/2.0;
}
// --- mhdflux part 2
mhdflux(v,&thread_c,tvd1_u1,tvd1_b,&thread_n);
//
mhdflux_max[threadIdx.x]=thread_c;
__syncthreads();
if (threadIdx.x==0)
{
float temp_c_max;
temp_c_max=0.0;
for (int i=0; i<BLOCK_SIZE; i++)
{
        if (mhdflux_max[i]>temp_c_max) temp_c_max=mhdflux_max[i];
}
for (int i=0; i<BLOCK_SIZE; i++)
{
        s_c[i]=temp_c_max;
}
}
__syncthreads();
c=s_c[threadIdx.x];
//
if (c>0)
{
        for (int ii=0; ii<5; ii++)
        {
                v[ii]=v[ii]/c;
        }
}
// --- tvd1 part 2
for (int ii=0; ii<5; ii++)
{
	wr[ii]=tvd1_u1[ii]+v[ii];
}
for (int ii=0; ii<5; ii++)
{
	wl[ii]=tvd1_u1[ii]-v[ii];	
}
for (int ii=0; ii<5; ii++)
{
	fr[ii]=c*wr[ii];
}
for (int ii=0; ii<5; ii++)
{
	tvd1_s_tmp1[a2D_FinC(5,blockDim.x,ii,threadIdx.x)]=fr[ii];
}
__syncthreads();
float dfrp[5];
float dfrm[5];
float dfr[5];
for (int ii=0; ii<5; ii++)
{
	dfrp[ii]=(tvd1_s_tmp1[a2D_FinC(5,blockDim.x,ii,flu_imp)]-fr[ii])/2.0;
}
for (int ii=0; ii<5; ii++)
{
	dfrm[ii]=(fr[ii]-tvd1_s_tmp1[a2D_FinC(5,blockDim.x,ii,flu_imm)])/2.0;
}
for (int ii=0; ii<5; ii++)
{
	dfr[ii]=0;
}
//
__syncthreads();
for (int ii=0; ii<5; ii++)
{
	if (dfrp[ii]*dfrm[ii]>0) dfr[ii]=2.0*dfrp[ii]*dfrm[ii]/(dfrp[ii]+dfrm[ii]);
	
}
//
for (int ii=0; ii<5; ii++)
{
	tvd1_s_tmp2[a2D_FinC(5,blockDim.x,ii,threadIdx.x)]=wl[ii];
}
__syncthreads();
for (int ii=0; ii<5; ii++)
{
	fl[ii]=c*tvd1_s_tmp2[a2D_FinC(5,blockDim.x,ii,flu_imp)];
}
//
float dflp[5];
float dflm[5];
float dfl[5];
for (int ii=0; ii<5; ii++)
{
	tvd1_s_tmp1[a2D_FinC(5,blockDim.x,ii,threadIdx.x)]=fl[ii];
}
__syncthreads();
for (int ii=0; ii<5; ii++)
{
	dflp[ii]=(fl[ii]-tvd1_s_tmp1[a2D_FinC(5,blockDim.x,ii,flu_imp)])/2.0;
}
for (int ii=0; ii<5; ii++)
{
	dflm[ii]=(tvd1_s_tmp1[a2D_FinC(5,blockDim.x,ii,flu_imm)]-fl[ii])/2.0;
}
for (int ii=0; ii<5; ii++)
{
	dfl[ii]=0;
}
//
__syncthreads();
for (int ii=0; ii<5; ii++)
{
	if (dflp[ii]*dflm[ii]>0) dfl[ii]=2.0*dflp[ii]*dflm[ii]/(dflp[ii]+dflm[ii]);
}
//
for (int ii=0; ii<5; ii++)
{
	flux[ii]=(fr[ii]-fl[ii]+(dfr[ii]-dfl[ii]))/2.0;
}
//
for (int ii=0; ii<5; ii++)
{
	tvd1_s_tmp2[a2D_FinC(5,blockDim.x,ii,threadIdx.x)]=flux[ii];
}
__syncthreads();
for (int ii=0; ii<5; ii++)
{
	flu_s_u[a2D_FinC(5,blockDim.x,ii,threadIdx.x)]=flu_s_u[a2D_FinC(5,blockDim.x,ii,threadIdx.x)]-(flux[ii]-tvd1_s_tmp2[a2D_FinC(5,blockDim.x,ii,flu_imm)])*(*flu_dt);
}
for (int ii=0; ii<5; ii++)
{
	flu_u[a4D_FinC(5,blockDim.x,gridDim.x,gridDim.y,ii,threadIdx.x,blockIdx.x,blockIdx.y)]=flu_s_u[a2D_FinC(5,blockDim.x,ii,threadIdx.x)];
}
// --- end tvd1
return;
}


__device__ void mhdflux(float *mf_v, float *mf_c, float *mf_u, float *mf_b, int *mf_n)
{
float gamma;
gamma=5.0/3.0;
//
float vx;
vx=mf_u[(2-1)]/mf_u[(1-1)];
//
float ps;
ps=(mf_u[(5-1)]-(mf_u[(2-1)]*mf_u[(2-1)]+mf_u[(3-1)]*mf_u[(3-1)]+mf_u[(4-1)]*mf_u[(4-1)])/mf_u[(1-1)]/2.0)*(gamma-1.0)+(2.0-gamma)*(mf_b[(1-1)]*mf_b[(1-1)]+mf_b[(2-1)]*mf_b[(2-1)]+mf_b[(3-1)]*mf_b[(3-1)])/2.0;
//
mf_v[(1-1)]=mf_u[(2-1)];
mf_v[(2-1)]=mf_u[(2-1)]*vx+ps-mf_b[(1-1)]*mf_b[(1-1)];
mf_v[(3-1)]=mf_u[(3-1)]*vx-mf_b[(2-1)]*mf_b[(1-1)];
mf_v[(4-1)]=mf_u[(4-1)]*vx-mf_b[(3-1)]*mf_b[(1-1)];
mf_v[(5-1)]=(mf_u[(5-1)]+ps)*vx-mf_b[(1-1)]*(mf_b[(1-1)]*mf_u[(2-1)]+mf_b[(2-1)]*mf_u[(3-1)]+mf_b[(3-1)]*mf_u[(4-1)])/mf_u[(1-1)];
//
float p;
p=ps-(mf_b[(1-1)]*mf_b[(1-1)]+mf_b[(2-1)]*mf_b[(2-1)]+mf_b[(3-1)]*mf_b[(3-1)])/2.0;
//
(*mf_c)=fabs(vx)+sqrt(fabs((mf_b[(1-1)]*mf_b[(1-1)]+mf_b[(2-1)]*mf_b[(2-1)]+mf_b[(3-1)]*mf_b[(3-1)]+gamma*p)/mf_u[(1-1)]));
//
return;
}

