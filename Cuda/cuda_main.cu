#include <stdio.h>
#include <math.h>
#include "hip/hip_runtime.h"

#include "parameter.h" 
#include "array_definition.h"
#include "cuda_funclist.h"
#include "cuda_function.h" 
#include "cuda_subroutine.h"

extern "C" void cuda_main(float *h_u, float *h_b, int *h_nx, int *h_ny, int *h_nz)
{
//	general info initialization
int Totalthreads = (*h_nx)*(*h_ny)*(*h_nz);
int numThreadsPerBlock = *h_nx;
int numBlocks = Totalthreads/numThreadsPerBlock;
int NumOfU = 5;
int NumOfB = 3;
//	memory size initialization
size_t u_memSize = NumOfU * numBlocks * numThreadsPerBlock * sizeof(float);
size_t b_memSize = NumOfB * numBlocks * numThreadsPerBlock * sizeof(float);
size_t c_memSize = numBlocks * numThreadsPerBlock * sizeof(float);
size_t int_memSize = sizeof(int);
size_t float_memSize = sizeof(float);
//	data on the host
float *h_dt;
//	data on the device
//	hipMalloc
//	for general purpose
float *d_u, *d_b;
hipMalloc( (void **) &d_u, u_memSize );
hipMalloc( (void **) &d_b, b_memSize );
int *d_nx,*d_ny,*d_nz;
hipMalloc( (void **) &d_nx, int_memSize );
hipMalloc( (void **) &d_ny, int_memSize );
hipMalloc( (void **) &d_nz, int_memSize );
float *d_dt;
hipMalloc( (void **) &d_dt, float_memSize );
//	for cuda_cfl
float *d_c;
hipMalloc( (void **) &d_c, c_memSize );
//	for cuda_advectbyzx
float *d_temp;
hipMalloc( (void **) &d_temp, c_memSize );
//	for cuda_transpose
float *d_ut, *d_bt;
hipMalloc( (void **) &d_ut, u_memSize );
hipMalloc( (void **) &d_bt, b_memSize );
//	hipMemcpy
//	copy data from host to device
hipMemcpy( d_u, h_u, u_memSize, hipMemcpyHostToDevice );
hipMemcpy( d_b, h_b, b_memSize, hipMemcpyHostToDevice );
hipMemcpy( d_nx, h_nx, int_memSize, hipMemcpyHostToDevice );
hipMemcpy( d_ny, h_ny, int_memSize, hipMemcpyHostToDevice );
hipMemcpy( d_nz, h_nz, int_memSize, hipMemcpyHostToDevice );
//
checkCUDAError("memcpy: from host to device, in cuda_main");
//	initialize data for loop
float t,dt,tf;
int iter;
float ct;
t=0;
iter=0;
ct=100.;
tf=ct*10;
//	initialization for timing
hipEvent_t start, stop;
hipEventCreate(&start);
hipEventCreate(&stop);
//	in milliseconds with a resolution of around 0.5 microseconds
float elapsedTime;
do {
//	start the timer
	hipEventRecord(start,0);
//	output
//	if you want to output, you have to use hipMemcpy
//	copy the data from device to host to output
	hipMemcpy( h_u, d_u, u_memSize, hipMemcpyDeviceToHost );
	hipMemcpy( h_b, d_b, b_memSize, hipMemcpyDeviceToHost );
	printf("t=	%f,	%i,	%f\n",t,iter,h_u[a4D_FinC(5,(*h_nx),(*h_ny),(*h_nz),(5-1),(*h_nx)/4,1,1)]);
//	done output
	iter=iter+1;
	cuda_cfl(d_u,d_b,d_nx,d_ny,d_nz,d_dt,d_c,h_nx,h_ny,h_nz,h_dt);
	dt=0.9*(*h_dt);
	//dt=0.5;
	if (dt>(tf-t)/2.0) dt=(tf-t)/2.0;
	t=t+2.0*dt;
//	start sweep
	cuda_fluidx(d_u,d_b,d_nx,d_ny,d_nz,d_dt,h_nx,h_ny,h_nz);
	cuda_advectbyzx(d_u,d_b,d_nx,d_ny,d_nz,d_dt,d_temp,h_nx,h_ny,h_nz);
//	the y sweep
	cuda_transpose12(d_ut,d_bt,d_u,d_b,d_nx,d_ny,d_nz,h_nx,h_ny,h_nz);
	cuda_fluidx(d_u,d_b,d_ny,d_nx,d_nz,d_dt,h_ny,h_nx,h_nz);
	cuda_advectbyzx(d_u,d_b,d_ny,d_nx,d_nz,d_dt,d_temp,h_ny,h_nx,h_nz);
//	z sweep
	cuda_transpose13(d_ut,d_bt,d_u,d_b,d_ny,d_nx,d_nz,h_ny,h_nx,h_nz);
	cuda_fluidx(d_u,d_b,d_nz,d_nx,d_ny,d_dt,h_nz,h_nx,h_ny);
	cuda_advectbyzx(d_u,d_b,d_nz,d_nx,d_ny,d_dt,d_temp,h_nz,h_nx,h_ny);
	cuda_advectbyzx(d_u,d_b,d_nz,d_nx,d_ny,d_dt,d_temp,h_nz,h_nx,h_ny);
	cuda_fluidx(d_u,d_b,d_nz,d_nx,d_ny,d_dt,h_nz,h_nx,h_ny);

//	back
	cuda_transpose13(d_ut,d_bt,d_u,d_b,d_nz,d_nx,d_ny,h_nz,h_nx,h_ny);
	cuda_advectbyzx(d_u,d_b,d_ny,d_nx,d_nz,d_dt,d_temp,h_ny,h_nx,h_nz);
	cuda_fluidx(d_u,d_b,d_ny,d_nx,d_nz,d_dt,h_ny,h_nx,h_nz);
//	x again
	cuda_transpose12(d_ut,d_bt,d_u,d_b,d_ny,d_nx,d_nz,h_ny,h_nx,h_nz);
	cuda_advectbyzx(d_u,d_b,d_nx,d_ny,d_nz,d_dt,d_temp,h_nx,h_ny,h_nz);
	cuda_fluidx(d_u,d_b,d_nx,d_ny,d_nz,d_dt,h_nx,h_ny,h_nz);
//	finish sweep
//	stop the timer
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime,start,stop);	
	printf("time per loop(in milliseconds):	%f\n",elapsedTime);
} while (t<tf);
//
//      hipMemcpy
//      copy data from device to host
hipMemcpy( h_u, d_u, u_memSize, hipMemcpyDeviceToHost );
hipMemcpy( h_b, d_b, b_memSize, hipMemcpyDeviceToHost );
//
checkCUDAError("memcpy: from device to host, in cuda_main");
//
hipFree(d_u);
hipFree(d_b);
hipFree(d_nx);
hipFree(d_ny);
hipFree(d_nz);
hipFree(d_dt);
//
hipEventDestroy(start);
hipEventDestroy(stop);
//
return;
}

