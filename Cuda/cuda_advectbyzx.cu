#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "hip/hip_runtime.h"

#include "parameter.h"
#include "array_definition.h" 
#include "cuda_funclist.h"

//      advectbyzx
__global__ void advectbyzx1_kernel(float *adv1_u, float *adv1_b, int *adv1_nx, int *adv1_ny, int *adv1_nz, float *adv1_dt, float *adv1_temp);
__global__ void advectbyzx2_kernel(float *adv2_u, float *adv2_b, int *adv2_nx, int *adv2_ny, int *adv2_nz, float *adv2_dt, float *adv2_temp);
__global__ void advectbyzx1b_kernel(float *adv1_u, float *adv1_b, int *adv1_nx, int *adv1_ny, int *adv1_nz, float *adv1_dt, float *adv1_temp);
__global__ void advectbyzx2b_kernel(float *adv2_u, float *adv2_b, int *adv2_nx, int *adv2_ny, int *adv2_nz, float *adv2_dt, float *adv2_temp);

void cuda_advectbyzx(float *adv_u, float *adv_b, int *adv_nx, int *adv_ny, int *adv_nz, float *adv_dt, float *adv_temp, int *h_adv_nx, int *h_adv_ny, int *h_adv_nz)
{
//      send it to device to calculate
dim3 dimGrid(*h_adv_ny,*h_adv_nz);
dim3 dimBlock(*h_adv_nx);
advectbyzx1_kernel<<< dimGrid, dimBlock >>>(adv_u,adv_b,adv_nx,adv_ny,adv_nz,adv_dt,adv_temp);
advectbyzx1b_kernel<<< dimGrid, dimBlock >>>(adv_u,adv_b,adv_nx,adv_ny,adv_nz,adv_dt,adv_temp);
advectbyzx2_kernel<<< dimGrid, dimBlock >>>(adv_u,adv_b,adv_nx,adv_ny,adv_nz,adv_dt,adv_temp);
advectbyzx2b_kernel<<< dimGrid, dimBlock >>>(adv_u,adv_b,adv_nx,adv_ny,adv_nz,adv_dt,adv_temp);
//
hipDeviceSynchronize();
//
checkCUDAError("kernel execution in cuda_advectbyzx");
//
}

//      
__global__ void advectbyzx1_kernel(float *adv1_u, float *adv1_b, int *adv1_nx, int *adv1_ny, int *adv1_nz, float *adv1_dt, float *adv1_temp)
{
/*
two dimensional array of blocks on grid where each block has one dimensional array of threads:
UniqueBlockIndex = blockIdx.y * gridDim.x + blockIdx.x;
UniqueThreadIndex = UniqueBlockIndex * blockDim.x + threadIdx.x;
*/
/*
i = threadIdx.x
j = blockIdx.x
k = blockIdx.y
nx = blockDim.x
ny = gridDim.x
nz = gridDim.y
*/
//
__shared__ float adv1_s_u[5*BLOCK_SIZE];
__shared__ float adv1_s_u_jm[5*BLOCK_SIZE];
__shared__ float adv1_s_b[3*BLOCK_SIZE];
//
int adv1_jm;
adv1_jm=(blockIdx.x+(*adv1_ny)-1)%(*adv1_ny);
//
for (int ii=0; ii<5; ii++)
{
	adv1_s_u[a2D_FinC(5,blockDim.x,ii,threadIdx.x)]=adv1_u[a4D_FinC(5,blockDim.x,gridDim.x,gridDim.y,ii,threadIdx.x,blockIdx.x,blockIdx.y)];
}
for (int ii=0; ii<5; ii++)
{
        adv1_s_u_jm[a2D_FinC(5,blockDim.x,ii,threadIdx.x)]=adv1_u[a4D_FinC(5,blockDim.x,gridDim.x,gridDim.y,ii,threadIdx.x,adv1_jm,blockIdx.y)];
}
for (int ii=0; ii<3; ii++)
{
        adv1_s_b[a2D_FinC(3,blockDim.x,ii,threadIdx.x)]=adv1_b[a4D_FinC(3,blockDim.x,gridDim.x,gridDim.y,ii,threadIdx.x,blockIdx.x,blockIdx.y)];
}
__syncthreads();
//
float vx;
vx=(adv1_s_u_jm[a2D_FinC(5,blockDim.x,(2-1),threadIdx.x)]+adv1_s_u[a2D_FinC(5,blockDim.x,(2-1),threadIdx.x)])/(adv1_s_u_jm[a2D_FinC(5,blockDim.x,(1-1),threadIdx.x)]+adv1_s_u[a2D_FinC(5,blockDim.x,(1-1),threadIdx.x)]);
//
int adv1_imm,adv1_imp;
adv1_imm=(threadIdx.x+(*adv1_nx)-1)%(*adv1_nx);
adv1_imp=(threadIdx.x+1)%(*adv1_nx);
//
__shared__ float adv1_s_tmp1[BLOCK_SIZE];
adv1_s_tmp1[threadIdx.x]=vx;
__syncthreads();
//
vx=(adv1_s_tmp1[adv1_imm]+adv1_s_tmp1[adv1_imp]+2.0*adv1_s_tmp1[threadIdx.x])/4.0;
//
float b1x;
b1x=adv1_s_b[a2D_FinC(3,blockDim.x,(2-1),threadIdx.x)];
//
//      first tvdb
float vg;
vg=vx;
float b;
b=b1x;
__shared__ float adv1_s_vg[BLOCK_SIZE];
adv1_s_vg[threadIdx.x]=vx;
__syncthreads();
//
float vh;
vh=(adv1_s_vg[threadIdx.x]+adv1_s_vg[adv1_imp])/2.0;
//
__shared__ float adv1_s_tmp2[BLOCK_SIZE];
adv1_s_tmp2[threadIdx.x]=b*vg;
__syncthreads();
float flux1;
if (vh>0) flux1=b*vg;
else flux1=adv1_s_tmp2[adv1_imp];
adv1_s_tmp1[threadIdx.x]=flux1;
__syncthreads();
float b1;
b1=b-(flux1-adv1_s_tmp1[adv1_imm])*(*adv1_dt)/2.0;
//
int ip;
int ipp;
int im;
ip=(threadIdx.x+1)%(*adv1_nx);
ipp=(ip+1)%(*adv1_nx);
im=(threadIdx.x+(*adv1_nx)-1)%(*adv1_nx);
//
float v;
v=vh;
float w;
float wp;
float wm;
__shared__ float adv1_s_b1_tvdb[BLOCK_SIZE];
adv1_s_b1_tvdb[threadIdx.x]=b1;
__syncthreads();
if (v>0)
{
        w=adv1_s_vg[threadIdx.x]*adv1_s_b1_tvdb[threadIdx.x];
        wp=(adv1_s_vg[ip]*adv1_s_b1_tvdb[ip]-w)/2.0;
        wm=(w-adv1_s_vg[im]*adv1_s_b1_tvdb[im])/2.0;
}
else
{
        w=adv1_s_vg[ip]*adv1_s_b1_tvdb[ip];
        wp=(w-adv1_s_vg[ipp]*adv1_s_b1_tvdb[ipp])/2.0;
        wm=(adv1_s_vg[threadIdx.x]*adv1_s_b1_tvdb[threadIdx.x]-w)/2.0;
}
float dw;
dw=0.0;
//
if (wm*wp>0) dw=2.0*wm*wp/(wm+wp);
float flux;
flux=(w+dw)*(*adv1_dt);
//
adv1_s_tmp2[threadIdx.x]=flux;
__syncthreads();
b=b-(flux-adv1_s_tmp2[adv1_imm]);
//      finished tvdb
//
adv1_s_b[a2D_FinC(3,blockDim.x,(2-1),threadIdx.x)]=b;
adv1_s_b[a2D_FinC(3,blockDim.x,(1-1),threadIdx.x)]=adv1_s_b[a2D_FinC(3,blockDim.x,(1-1),threadIdx.x)]-adv1_s_tmp2[adv1_imm];
//
//	send it back to global
for (int ii=0; ii<3; ii++)
{
	adv1_b[a4D_FinC(3,blockDim.x,gridDim.x,gridDim.y,ii,threadIdx.x,blockIdx.x,blockIdx.y)]=adv1_s_b[a2D_FinC(3,blockDim.x,ii,threadIdx.x)];
}
adv1_temp[a3D_FinC(blockDim.x,gridDim.x,gridDim.y,threadIdx.x,blockIdx.x,blockIdx.y)]=adv1_s_tmp2[adv1_imm];
//
return;
}

__global__ void advectbyzx1b_kernel(float *adv1_u, float *adv1_b, int *adv1_nx, int *adv1_ny, int *adv1_nz, float *adv1_dt, float *adv1_temp)
{
/*
two dimensional array of blocks on grid where each block has one dimensional array of threads:
UniqueBlockIndex = blockIdx.y * gridDim.x + blockIdx.x;
UniqueThreadIndex = UniqueBlockIndex * blockDim.x + threadIdx.x;
*/
/*
i = threadIdx.x
j = blockIdx.x
k = blockIdx.y
nx = blockDim.x
ny = gridDim.x
nz = gridDim.y
*/
int adv1_jm;
adv1_jm=(blockIdx.x+(*adv1_ny)-1)%(*adv1_ny);
//
adv1_b[a4D_FinC(3,blockDim.x,gridDim.x,gridDim.y,(1-1),threadIdx.x,adv1_jm,blockIdx.y)]=adv1_b[a4D_FinC(3,blockDim.x,gridDim.x,gridDim.y,(1-1),threadIdx.x,adv1_jm,blockIdx.y)]+adv1_temp[a3D_FinC(blockDim.x,gridDim.x,gridDim.y,threadIdx.x,blockIdx.x,blockIdx.y)];
//
return;
}

__global__ void advectbyzx2_kernel(float *adv2_u, float *adv2_b, int *adv2_nx, int *adv2_ny, int *adv2_nz, float *adv2_dt, float *adv2_temp)
{
/*
two dimensional array of blocks on grid where each block has one dimensional array of threads:
UniqueBlockIndex = blockIdx.y * gridDim.x + blockIdx.x;
UniqueThreadIndex = UniqueBlockIndex * blockDim.x + threadIdx.x;
*/
/*
i = threadIdx.x
j = blockIdx.x
k = blockIdx.y
nx = blockDim.x
ny = gridDim.x
nz = gridDim.y
*/
//
__shared__ float adv2_s_u[5*BLOCK_SIZE];
__shared__ float adv2_s_u_km[5*BLOCK_SIZE];
__shared__ float adv2_s_b[3*BLOCK_SIZE];
//
int adv2_km;
adv2_km=(blockIdx.y+(*adv2_nz)-1)%(*adv2_nz);
//
for (int ii=0; ii<5; ii++)
{
	adv2_s_u[a2D_FinC(5,blockDim.x,ii,threadIdx.x)]=adv2_u[a4D_FinC(5,blockDim.x,gridDim.x,gridDim.y,ii,threadIdx.x,blockIdx.x,blockIdx.y)];
}
for (int ii=0; ii<5; ii++)
{
        adv2_s_u_km[a2D_FinC(5,blockDim.x,ii,threadIdx.x)]=adv2_u[a4D_FinC(5,blockDim.x,gridDim.x,gridDim.y,ii,threadIdx.x,adv2_km,blockIdx.y)];
}
for (int ii=0; ii<3; ii++)
{
        adv2_s_b[a2D_FinC(3,blockDim.x,ii,threadIdx.x)]=adv2_b[a4D_FinC(3,blockDim.x,gridDim.x,gridDim.y,ii,threadIdx.x,blockIdx.x,blockIdx.y)];
}
__syncthreads();
//
float vx;
vx=(adv2_s_u_km[a2D_FinC(5,blockDim.x,(2-1),threadIdx.x)]+adv2_s_u[a2D_FinC(5,blockDim.x,(2-1),threadIdx.x)])/(adv2_s_u_km[a2D_FinC(5,blockDim.x,(1-1),threadIdx.x)]+adv2_s_u[a2D_FinC(5,blockDim.x,(1-1),threadIdx.x)]);
//
int adv2_imm,adv2_imp;
adv2_imm=(threadIdx.x+(*adv2_nx)-1)%(*adv2_nx);
adv2_imp=(threadIdx.x+1)%(*adv2_nx);
//
__shared__ float adv2_s_tmp1[BLOCK_SIZE];
adv2_s_tmp1[threadIdx.x]=vx;
__syncthreads();
//
vx=(adv2_s_tmp1[adv2_imm]+adv2_s_tmp1[adv2_imp]+2.0*adv2_s_tmp1[threadIdx.x])/4.0;
//
float b1x;
b1x=adv2_s_b[a2D_FinC(3,blockDim.x,(3-1),threadIdx.x)];
//
//      second tvdb
float vg;
vg=vx;
float b;
b=b1x;
__shared__ float adv2_s_vg[BLOCK_SIZE];
adv2_s_vg[threadIdx.x]=vx;
__syncthreads();
//
float vh;
vh=(adv2_s_vg[threadIdx.x]+adv2_s_vg[adv2_imp])/2.0;
//
__shared__ float adv2_s_tmp2[BLOCK_SIZE];
adv2_s_tmp2[threadIdx.x]=b*vg;
__syncthreads();
float flux1;
if (vh>0) flux1=b*vg;
else flux1=adv2_s_tmp2[adv2_imp];
adv2_s_tmp1[threadIdx.x]=flux1;
__syncthreads();
float b1;
b1=b-(flux1-adv2_s_tmp1[adv2_imm])*(*adv2_dt)/2.0;
//
int ip;
int ipp;
int im;
ip=(threadIdx.x+1)%(*adv2_nx);
ipp=(ip+1)%(*adv2_nx);
im=(threadIdx.x+(*adv2_nx)-1)%(*adv2_nx);
//
float v;
v=vh;
float w;
float wp;
float wm;
__shared__ float adv2_s_b1_tvdb[BLOCK_SIZE];
adv2_s_b1_tvdb[threadIdx.x]=b1;
__syncthreads();
if (v>0)
{
        w=adv2_s_vg[threadIdx.x]*adv2_s_b1_tvdb[threadIdx.x];
        wp=(adv2_s_vg[ip]*adv2_s_b1_tvdb[ip]-w)/2.0;
        wm=(w-adv2_s_vg[im]*adv2_s_b1_tvdb[im])/2.0;
}
else
{
        w=adv2_s_vg[ip]*adv2_s_b1_tvdb[ip];
        wp=(w-adv2_s_vg[ipp]*adv2_s_b1_tvdb[ipp])/2.0;
        wm=(adv2_s_vg[threadIdx.x]*adv2_s_b1_tvdb[threadIdx.x]-w)/2.0;
}
float dw;
dw=0.0;
//
if (wm*wp>0) dw=2.0*wm*wp/(wm+wp);
float flux;
flux=(w+dw)*(*adv2_dt);
//
adv2_s_tmp2[threadIdx.x]=flux;
__syncthreads();
b=b-(flux-adv2_s_tmp2[adv2_imm]);
//      finished tvdb
adv2_s_b[a2D_FinC(3,blockDim.x,(3-1),threadIdx.x)]=b;
adv2_s_b[a2D_FinC(3,blockDim.x,(1-1),threadIdx.x)]=adv2_s_b[a2D_FinC(3,blockDim.x,(1-1),threadIdx.x)]-adv2_s_tmp2[adv2_imm];
for (int ii=0; ii<3; ii++)
{
	adv2_b[a4D_FinC(3,blockDim.x,gridDim.x,gridDim.y,ii,threadIdx.x,blockIdx.x,blockIdx.y)]=adv2_s_b[a2D_FinC(3,blockDim.x,ii,threadIdx.x)];
}
adv2_temp[a3D_FinC(blockDim.x,gridDim.x,gridDim.y,threadIdx.x,blockIdx.x,blockIdx.y)]=adv2_s_tmp2[adv2_imm];
//
return;
}

__global__ void advectbyzx2b_kernel(float *adv2_u, float *adv2_b, int *adv2_nx, int *adv2_ny, int *adv2_nz, float *adv2_dt, float *adv2_temp)
{
/*
two dimensional array of blocks on grid where each block has one dimensional array of threads:
UniqueBlockIndex = blockIdx.y * gridDim.x + blockIdx.x;
UniqueThreadIndex = UniqueBlockIndex * blockDim.x + threadIdx.x;
*/
/*
i = threadIdx.x
j = blockIdx.x
k = blockIdx.y
nx = blockDim.x
ny = gridDim.x
nz = gridDim.y
*/
//
int adv2_km;
adv2_km=(blockIdx.y+(*adv2_nz)-1)%(*adv2_nz);
//
adv2_b[a4D_FinC(3,blockDim.x,gridDim.x,gridDim.y,(1-1),threadIdx.x,blockIdx.x,adv2_km)]=adv2_b[a4D_FinC(3,blockDim.x,gridDim.x,gridDim.y,(1-1),threadIdx.x,blockIdx.x,adv2_km)]+adv2_temp[a3D_FinC(blockDim.x,gridDim.x,gridDim.y,threadIdx.x,blockIdx.x,blockIdx.y)];
//
return;
}
