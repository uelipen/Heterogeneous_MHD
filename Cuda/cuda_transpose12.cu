#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "hip/hip_runtime.h"

#include "parameter.h" 
#include "array_definition.h"
#include "cuda_funclist.h"

__global__ void transpose12_kernel(float *t12_ut, float *t12_bt, float *t12_u, float *t12_b, int *t12_nx, int *t12_ny, int *t12_nz);
//	

void cuda_transpose12(float *t12_ut, float *t12_bt, float *t12_u, float *t12_b, int *t12_nx, int *t12_ny, int *t12_nz, int *h_t12_nx, int *h_t12_ny, int *h_t12_nz)
{
//      initialization
int Totalthreads = (*h_t12_nx)*(*h_t12_ny)*(*h_t12_nz);
int numThreadsPerBlock = *h_t12_nx;
int numBlocks = Totalthreads/numThreadsPerBlock;
int NumOfU = 5;
int NumOfB = 3;
size_t u_memSize = NumOfU * numBlocks * numThreadsPerBlock * sizeof(float);
size_t b_memSize = NumOfB * numBlocks * numThreadsPerBlock * sizeof(float);
//      send it to device to calculate
dim3 dimGrid(*h_t12_ny,*h_t12_nz);
dim3 dimBlock(*h_t12_nx);
transpose12_kernel<<< dimGrid, dimBlock >>>(t12_ut,t12_bt,t12_u,t12_b,t12_nx,t12_ny,t12_nz);
//
hipDeviceSynchronize();
//
checkCUDAError("kernel execution in cuda_transpose12");
//	hipMemcpy
//	from d_ut to d_u, in device
//	from d_bt to d_b, in device
hipMemcpy(t12_u,t12_ut, u_memSize, hipMemcpyDeviceToDevice );
hipMemcpy(t12_b,t12_bt, b_memSize, hipMemcpyDeviceToDevice );
//
checkCUDAError("memcpy: from device to device, in cuda_transpose12");
//
}

__global__ void transpose12_kernel(float *t12_ut, float *t12_bt, float *t12_u, float *t12_b, int *t12_nx, int *t12_ny, int *t12_nz)
{
/*
two dimensional array of blocks on grid where each block has one dimensional array of threads:
UniqueBlockIndex = blockIdx.y * gridDim.x + blockIdx.x;
UniqueThreadIndex = UniqueBlockIndex * blockDim.x + threadIdx.x;
*/
/*
i = threadIdx.x
j = blockIdx.x
k = blockIdx.y
nx = blockDim.x
ny = gridDim.x
nz = gridDim.y
*/
t12_ut[a4D_FinC(5,gridDim.x,blockDim.x,gridDim.y,(1-1),blockIdx.x,threadIdx.x,blockIdx.y)]=t12_u[a4D_FinC(5,blockDim.x,gridDim.x,gridDim.y,(1-1),threadIdx.x,blockIdx.x,blockIdx.y)];
t12_ut[a4D_FinC(5,gridDim.x,blockDim.x,gridDim.y,(2-1),blockIdx.x,threadIdx.x,blockIdx.y)]=t12_u[a4D_FinC(5,blockDim.x,gridDim.x,gridDim.y,(3-1),threadIdx.x,blockIdx.x,blockIdx.y)];
t12_ut[a4D_FinC(5,gridDim.x,blockDim.x,gridDim.y,(3-1),blockIdx.x,threadIdx.x,blockIdx.y)]=t12_u[a4D_FinC(5,blockDim.x,gridDim.x,gridDim.y,(2-1),threadIdx.x,blockIdx.x,blockIdx.y)];
t12_ut[a4D_FinC(5,gridDim.x,blockDim.x,gridDim.y,(4-1),blockIdx.x,threadIdx.x,blockIdx.y)]=t12_u[a4D_FinC(5,blockDim.x,gridDim.x,gridDim.y,(4-1),threadIdx.x,blockIdx.x,blockIdx.y)];
t12_ut[a4D_FinC(5,gridDim.x,blockDim.x,gridDim.y,(5-1),blockIdx.x,threadIdx.x,blockIdx.y)]=t12_u[a4D_FinC(5,blockDim.x,gridDim.x,gridDim.y,(5-1),threadIdx.x,blockIdx.x,blockIdx.y)];
//
t12_bt[a4D_FinC(3,gridDim.x,blockDim.x,gridDim.y,(1-1),blockIdx.x,threadIdx.x,blockIdx.y)]=t12_b[a4D_FinC(3,blockDim.x,gridDim.x,gridDim.y,(2-1),threadIdx.x,blockIdx.x,blockIdx.y)];
t12_bt[a4D_FinC(3,gridDim.x,blockDim.x,gridDim.y,(2-1),blockIdx.x,threadIdx.x,blockIdx.y)]=t12_b[a4D_FinC(3,blockDim.x,gridDim.x,gridDim.y,(1-1),threadIdx.x,blockIdx.x,blockIdx.y)];
t12_bt[a4D_FinC(3,gridDim.x,blockDim.x,gridDim.y,(3-1),blockIdx.x,threadIdx.x,blockIdx.y)]=t12_b[a4D_FinC(3,blockDim.x,gridDim.x,gridDim.y,(3-1),threadIdx.x,blockIdx.x,blockIdx.y)];
return;
}

//	
//-----------------------
/*
   Fortran subroutine arguments are passed by references.
   call fun( array_a, array_b, N) will be mapped to
   function (*a, *b, *N);
*/
extern "C" void cuda_transpose12_(float *h_ut, float *h_bt, float *h_u, float *h_b, int *h_nx, int *h_ny, int *h_nz, float *h_dt)
{
int Totalthreads = (*h_nx)*(*h_ny)*(*h_nz);
int numThreadsPerBlock = *h_nx;
int numBlocks = Totalthreads/numThreadsPerBlock;
int NumOfU = 5;
int NumOfB = 3;
//      intialize
size_t u_memSize = NumOfU * numBlocks * numThreadsPerBlock * sizeof(float);
size_t b_memSize = NumOfB * numBlocks * numThreadsPerBlock * sizeof(float);
//
float *d_u, *d_b;
hipMalloc( (void **) &d_u, u_memSize );
hipMalloc( (void **) &d_b, b_memSize );
hipMemcpy( d_u, h_u, u_memSize, hipMemcpyHostToDevice );
hipMemcpy( d_b, h_b, b_memSize, hipMemcpyHostToDevice );
//
float *d_ut, *d_bt;
hipMalloc( (void **) &d_ut, u_memSize );
hipMalloc( (void **) &d_bt, b_memSize );
//
int *d_nx,*d_ny,*d_nz;
size_t n_memSize = sizeof(int);
hipMalloc( (void **) &d_nx, n_memSize );
hipMalloc( (void **) &d_ny, n_memSize );
hipMalloc( (void **) &d_nz, n_memSize );
hipMemcpy( d_nx, h_nx, n_memSize, hipMemcpyHostToDevice );
hipMemcpy( d_ny, h_ny, n_memSize, hipMemcpyHostToDevice );
hipMemcpy( d_nz, h_nz, n_memSize, hipMemcpyHostToDevice );
//
dim3 dimGrid(*h_ny,*h_nz);
dim3 dimBlock(numThreadsPerBlock);
transpose12_kernel<<< dimGrid, dimBlock >>>( d_ut, d_bt, d_u, d_b, d_nx, d_ny, d_nz);
//
hipDeviceSynchronize();
//
checkCUDAError("kernel execution");
//
//	find the max
hipMemcpy( h_ut, d_ut, u_memSize, hipMemcpyDeviceToHost );
hipMemcpy( h_bt, d_bt, b_memSize, hipMemcpyDeviceToHost );
//
checkCUDAError("memcpy");
//
hipFree(d_u);
hipFree(d_b);
hipFree(d_nx);
hipFree(d_ny);
hipFree(d_nz);
hipFree(d_ut);
hipFree(d_bt);
//
return;
}



