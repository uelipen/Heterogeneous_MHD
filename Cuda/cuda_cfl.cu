#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "hip/hip_runtime.h"

#include "parameter.h" 
#include "array_definition.h"
#include "cuda_funclist.h"

__global__ void cfl_kernel(float *cfl_u, float *cfl_b, int *cfl_nx, int *cfl_ny, int *cfl_nz, float *cfl_dt, float *cfl_c);
__device__ float max_3num(float *m3_1, float *m3_2, float *m3_3);
__device__ float max_2num(float *m2_1, float *m2_2);
__host__ void h_cfl_find_max(float *hcfm_in, int *hcfm_ny, int *hcfm_nz, float *hcfm_out);

void cuda_cfl(float *cfl_u, float *cfl_b, int *cfl_nx, int *cfl_ny, int *cfl_nz, float *cfl_dt, float *cfl_c, int *h_cfl_nx, int *h_cfl_ny, int *h_cfl_nz, float *h_cfl_dt)
{
//	initialization
int Totalthreads = (*h_cfl_nx)*(*h_cfl_ny)*(*h_cfl_nz);
int numThreadsPerBlock = *h_cfl_nx;
int numBlocks = Totalthreads/numThreadsPerBlock;
size_t c_memSize = numBlocks * sizeof(float);
size_t dt_memSize = sizeof(float);
//	send it to device to calculate
dim3 dimGrid(*h_cfl_ny,*h_cfl_nz);
dim3 dimBlock(*h_cfl_nx);
cfl_kernel<<< dimGrid, dimBlock >>>( cfl_u, cfl_b, cfl_nx, cfl_ny, cfl_nz, cfl_dt, cfl_c);
//
hipDeviceSynchronize();
//
checkCUDAError("kernel execution in cuda_cfl");
//      get it from device to find the max of c
float *temp_h_cfl_c;
temp_h_cfl_c = (float *) malloc(c_memSize);
hipMemcpy( temp_h_cfl_c, cfl_c, c_memSize, hipMemcpyDeviceToHost );
//
checkCUDAError("memcpy: from device to host, in cuda_cfl");
//
float max_c;
max_c=0;
h_cfl_find_max(temp_h_cfl_c,h_cfl_ny,h_cfl_nz,&max_c);
//	find it and get cfl_dt in host
(*h_cfl_dt)=1/(max_c);
//	copy it to device
hipMemcpy( cfl_dt, h_cfl_dt, dt_memSize, hipMemcpyHostToDevice );
//
checkCUDAError("memcpy: from host to device, in cuda_cfl");
//
free(temp_h_cfl_c);
//
}

__global__ void cfl_kernel(float *cfl_u, float *cfl_b, int *cfl_nx, int *cfl_ny, int *cfl_nz, float *cfl_dt, float *cfl_c)
{
/*
two dimensional array of blocks on grid where each block has one dimensional array of threads:
UniqueBlockIndex = blockIdx.y * gridDim.x + blockIdx.x;
UniqueThreadIndex = UniqueBlockIndex * blockDim.x + threadIdx.x;
*/
__shared__ float cfl_s_b1_ip[BLOCK_SIZE];
__shared__ float cfl_s_b2_jp[BLOCK_SIZE];
__shared__ float cfl_s_b3_kp[BLOCK_SIZE];
__shared__ float cfl_s_b[3*BLOCK_SIZE];
__shared__ float cfl_s_u[5*BLOCK_SIZE];
/*
i = threadIdx.x
j = blockIdx.x
k = blockIdx.y
nx = blockDim.x
ny = gridDim.x
nz = gridDim.y
*/
float gamma;
gamma=5.0/3.0;
int ii;
int kp,jp,ip;
//      kp=mod(k,nz)+1
kp=(blockIdx.y+1)%(*cfl_nz);
//      jp=mod(j,ny)+1
jp=(blockIdx.x+1)%(*cfl_ny);
//      ip=mod(i,nx)+1
ip=(threadIdx.x+1)%(*cfl_nx);
//	get cfl_s_b1_ip
cfl_s_b1_ip[threadIdx.x]=cfl_b[a4D_FinC(3,blockDim.x,gridDim.x,gridDim.y,(1-1),ip,blockIdx.x,blockIdx.y)];
//	get cfl_s_b2_jp
cfl_s_b2_jp[threadIdx.x]=cfl_b[a4D_FinC(3,blockDim.x,gridDim.x,gridDim.y,(2-1),threadIdx.x,jp,blockIdx.y)];
//	get cfl_s_b3_kp
cfl_s_b3_kp[threadIdx.x]=cfl_b[a4D_FinC(3,blockDim.x,gridDim.x,gridDim.y,(3-1),threadIdx.x,blockIdx.x,kp)];
//	get cfl_s_u
for (ii=0;ii<5;ii++)
{
	cfl_s_u[a2D_FinC(5,blockDim.x,ii,threadIdx.x)]=cfl_u[a4D_FinC(5,blockDim.x,gridDim.x,gridDim.y,ii,threadIdx.x,blockIdx.x,blockIdx.y)];
}
//	get cfl_s_b
for (ii=0;ii<3;ii++)
{
	cfl_s_b[a2D_FinC(3,blockDim.x,ii,threadIdx.x)]=cfl_b[a4D_FinC(3,blockDim.x,gridDim.x,gridDim.y,ii,threadIdx.x,blockIdx.x,blockIdx.y)];
}
//
__syncthreads();
//
float bx,by,bz;
//	bx=(b(1,i,j,k)+b(1,ip,j,k))/2
bx=(cfl_s_b[a2D_FinC(3,blockDim.x,(1-1),threadIdx.x)]+cfl_s_b1_ip[threadIdx.x])/2.0;
//	by=(b(2,i,j,k)+b(2,i,jp,k))/2
by=(cfl_s_b[a2D_FinC(3,blockDim.x,(2-1),threadIdx.x)]+cfl_s_b2_jp[threadIdx.x])/2.0;
//	bz=(b(3,i,j,k)+b(3,i,j,kp))/2
bz=(cfl_s_b[a2D_FinC(3,blockDim.x,(3-1),threadIdx.x)]+cfl_s_b3_kp[threadIdx.x])/2.0;
float v;
//	v=maxval(abs(u(2:4,i,j,k)/u(1,i,j,k)))
float temp1,temp2,temp3;
temp1=fabs(cfl_s_u[a2D_FinC(5,blockDim.x,(2-1),threadIdx.x)]/cfl_s_u[a2D_FinC(5,blockDim.x,(1-1),threadIdx.x)]);
temp2=fabs(cfl_s_u[a2D_FinC(5,blockDim.x,(3-1),threadIdx.x)]/cfl_s_u[a2D_FinC(5,blockDim.x,(1-1),threadIdx.x)]);
temp3=fabs(cfl_s_u[a2D_FinC(5,blockDim.x,(4-1),threadIdx.x)]/cfl_s_u[a2D_FinC(5,blockDim.x,(1-1),threadIdx.x)]);
v=max_3num(&temp1,&temp2,&temp3);
float b2;
b2=bx*bx+by*by+bz*bz;
//	ps=(u(5,i,j,k)-sum(u(2:4,i,j,k)**2,1)/u(1,i,j,k)/2)*(gamma-1)+(2-gamma)*b2/2
float ps;
ps=(cfl_s_u[a2D_FinC(5,blockDim.x,(5-1),threadIdx.x)]-(cfl_s_u[a2D_FinC(5,blockDim.x,(2-1),threadIdx.x)]*cfl_s_u[a2D_FinC(5,blockDim.x,(2-1),threadIdx.x)]+cfl_s_u[a2D_FinC(5,blockDim.x,(3-1),threadIdx.x)]*cfl_s_u[a2D_FinC(5,blockDim.x,(3-1),threadIdx.x)]+cfl_s_u[a2D_FinC(5,blockDim.x,(4-1),threadIdx.x)]*cfl_s_u[a2D_FinC(5,blockDim.x,(4-1),threadIdx.x)])/cfl_s_u[a2D_FinC(5,blockDim.x,(1-1),threadIdx.x)]/2.0)*(gamma-1.0)+(2.0-gamma)*b2/2.0;
//	p=ps-b2/2
float p;
p=ps-b2/2.0;
//	c=max(c,v+sqrt(abs(  (b2*2+gamma*p)/u(1,i,j,k))))
temp1=v+sqrt(fabs((b2*2.0+gamma*p)/cfl_s_u[a2D_FinC(5,blockDim.x,(1-1),threadIdx.x)]));
//	find max
__shared__ float cfl_s_c[BLOCK_SIZE];
float temp_c_max;
cfl_s_c[threadIdx.x]=temp1;
__syncthreads();
if (threadIdx.x==0)
{
temp_c_max=0.0;
for (int i=0; i<BLOCK_SIZE; i++)
{
        if (cfl_s_c[i]>temp_c_max) temp_c_max=cfl_s_c[i];
}
(*cfl_c)=temp_c_max;
}
//
return;
}

__device__ float max_3num(float *m3_1, float *m3_2, float *m3_3)
{
if ((*m3_1)>(*m3_2))
{
        if ((*m3_1)>(*m3_3))
        {
                return (*m3_1);
        }
        else
        {
                return (*m3_3);
        }
}
else
{
        if ((*m3_2)>(*m3_3))
        {
                return (*m3_2);
        }
        else
        {
                return (*m3_3);
        }
}
}

__device__ float max_2num(float *m2_1, float *m2_2)
{
if ((*m2_1)>(*m2_2))
{
        return (*m2_1);
}
else
{
        return (*m2_2);
}
}

__host__ void h_cfl_find_max(float *hcfm_in, int *hcfm_ny, int *hcfm_nz, float *hcfm_out)
{
int j,k;
(*hcfm_out)=0;
for (k=0;k<(*hcfm_nz);k++)
{
	for (j=0;j<(*hcfm_ny);j++)
	{
		if (hcfm_in[a2D_FinC((*hcfm_ny),(*hcfm_nz),j,k)]>(*hcfm_out))
		{
			(*hcfm_out)=hcfm_in[a2D_FinC((*hcfm_ny),(*hcfm_nz),j,k)];
		}		
	}
}
}

